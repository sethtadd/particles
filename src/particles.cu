#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <glad/gl.h> // Include first to avoid errors
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "Shader.hpp"
#include "CudaKernels.hpp"

const uint WIDTH = 828;
const uint HEIGHT = 512;

void glfwErrorCallback(int error, const char *description);

void handleInput(GLFWwindow *window);

// Callback functions
void mouse_callback(GLFWwindow *window, double xPos, double yPos);               // Mouse movement
void scroll_callback(GLFWwindow *window, double xOffset, double yOffset);        // Zooming in/out
void framebuffer_size_callback(GLFWwindow *window, int newWidth, int newHeight); // Handle window resizing

__global__ void updateParticles(float3 *d_instanceData, int numParticles, float time)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles)
    {
        d_instanceData[i].x += cos(2 * time + i) * 0.005f;
        d_instanceData[i].y += sin(2 * time + i) * 0.005f;
    }
}

int main()
{
    glfwSetErrorCallback(glfwErrorCallback);
    if (!glfwInit())
        return -1;

    GLFWwindow *window = glfwCreateWindow(WIDTH, HEIGHT, "Hello World", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    // set callback functions
    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetScrollCallback(window, scroll_callback);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // Initialize GLAD2
    int version = gladLoadGL(glfwGetProcAddress);

    // Print version info
    std::cout << "GLAD2 GL version: " << GLAD_VERSION_MAJOR(version) << "." << GLAD_VERSION_MINOR(version) << std::endl;
    std::cout << "OpenGL version: " << glGetString(GL_VERSION) << std::endl;
    std::cout << "Renderer: " << glGetString(GL_RENDERER) << std::endl;
    std::cout << "Vendor: " << glGetString(GL_VENDOR) << std::endl;

    glViewport(0, 0, WIDTH, HEIGHT);

    Shader shader("shaders/particles.vertex.glsl", "shaders/particles.geometry.glsl", "shaders/particles.fragment.glsl");

    float vertices[] = {
        0.0f, 0.0f, 0.0f,      // Position (x, y, z)
        1.0f, 1.0f, 1.0f, 1.0f // Color (r, g, b, a)
    };

    // Particle offsets
    float instanceData[] = {
        0.5f, 0.5f, 0.0f,   // Top right
        0.5f, -0.5f, 0.0f,  // Bottom right
        -0.5f, -0.5f, 0.0f, // Bottom left
        -0.5f, 0.5f, 0.0f   // Top left
    };
    int numParticles = sizeof(instanceData) / (3 * sizeof(float));
    std::cout << "Particle count: " << numParticles << std::endl;

    // Instance particles VAO
    GLuint vao;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);

    // Singular particle vertex data
    GLuint particleVbo;
    glGenBuffers(1, &particleVbo);
    glBindBuffer(GL_ARRAY_BUFFER, particleVbo);
    glBufferData(
        GL_ARRAY_BUFFER,  // target
        sizeof(vertices), // size
        vertices,         // data
        GL_STATIC_DRAW);  // usage

    // Position attribute
    glVertexAttribPointer(
        0,          // index
        3,          // size
        GL_FLOAT,   // type
        GL_FALSE,   // normalized
        0,          // stride
        (void *)0); // pointer
    glEnableVertexAttribArray(0);

    // Color attribute
    glVertexAttribPointer(
        1,                            // index
        4,                            // size
        GL_FLOAT,                     // type
        GL_FALSE,                     // normalized
        0,                            // stride
        (void *)(3 * sizeof(float))); // pointer
    glEnableVertexAttribArray(1);

    // Instance data
    GLuint instanceVbo;
    glGenBuffers(1, &instanceVbo);
    glBindBuffer(GL_ARRAY_BUFFER, instanceVbo);
    glBufferData(
        GL_ARRAY_BUFFER,      // target
        sizeof(instanceData), // size
        instanceData,         // data
        GL_STATIC_DRAW);      // usage

    // Offset attribute
    glVertexAttribPointer(
        2,                 // index
        3,                 // size
        GL_FLOAT,          // type
        GL_FALSE,          // normalized
        3 * sizeof(float), // stride
        (void *)0);        // pointer
    glEnableVertexAttribArray(2);
    glVertexAttribDivisor(2, 1); // Update attribute every 1 instance

    // Unbind particleVbo and vao
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);

    // Register instanceVbo with CUDA
    hipGraphicsResource *cuda_vbo_resource;
    hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, instanceVbo, cudaGraphicsMapFlagsWriteDiscard);

    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    while (!glfwWindowShouldClose(window))
    {
        glClearColor(0.533f, 0.438f, 0.723f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // Map instanceVbo to CUDA
        float3 *d_instanceData;
        hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
        size_t num_bytes;
        hipGraphicsResourceGetMappedPointer((void **)&d_instanceData, &num_bytes, cuda_vbo_resource);

        // Update particles
        int threadsPerBlock = 256;
        int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
        updateParticles<<<blocksPerGrid, threadsPerBlock>>>(d_instanceData, numParticles, glfwGetTime());
        hipError_t err = hipDeviceSynchronize();
        err = hipDeviceSynchronize(); // Blocks execution until kernel is finished
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to synchronize on the CUDA device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Unmap instanceVbo from CUDA (so OpenGL can use it)
        hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);

        // Draw particles
        shader.use();
        glBindVertexArray(vao);
        glDrawArraysInstanced(
            GL_POINTS,     // mode: type of primitives to render
            0,             // first: starting index
            1,             // count: number of indicies to render
            numParticles); // instance count
        glBindVertexArray(0);

        handleInput(window);
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Clean up
    hipGraphicsUnregisterResource(cuda_vbo_resource);
    glDeleteVertexArrays(1, &vao);
    glDeleteBuffers(1, &particleVbo);
    glDeleteBuffers(1, &instanceVbo);
    glfwTerminate();
    return 0;
}

void glfwErrorCallback(int error, const char *description)
{
    std::cerr << "GLFW Error: " << error << " - " << description << std::endl;
}

void handleInput(GLFWwindow *window)
{
    // exit program
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
    {
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    }
}

void mouse_callback(GLFWwindow *window, double xPos, double yPos)
{
    // TODO Implement
    // std::cout << "Mouse position: (" << xPos << ", " << yPos << ")" << std::endl;
}

void scroll_callback(GLFWwindow *window, double xOffset, double yOffset)
{
    std::cout << "Scroll offset: (" << xOffset << ", " << yOffset << ")" << std::endl;
}

void framebuffer_size_callback(GLFWwindow *window, int newWidth, int newHeight)
{
    glViewport(0, 0, newWidth, newHeight);
}
