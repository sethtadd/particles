#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <glad/gl.h> // Include first to avoid errors
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "Shader.hpp"
#include "CudaHelpers.cuh"

const uint WIDTH = 1024;
const uint HEIGHT = 1024;

void glfwErrorCallback(int error, const char *description);

void handleInput(GLFWwindow *window);

// Callback functions
void mouse_callback(GLFWwindow *window, double xPos, double yPos);               // Mouse movement
void scroll_callback(GLFWwindow *window, double xOffset, double yOffset);        // Zooming in/out
void framebuffer_size_callback(GLFWwindow *window, int newWidth, int newHeight); // Handle window resizing

__global__ void updateParticles(float3 *d_instancePositions, float3 *d_instanceVelocities, float4 *d_instanceColors, int numParticles, float time)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles)
    {
        d_instancePositions[i].x += d_instanceVelocities[i].x;
        d_instancePositions[i].y += d_instanceVelocities[i].y;

        if (d_instancePositions[i].x > 1.0f)
        {
            d_instancePositions[i].x = 1.0f;
            d_instanceVelocities[i].x *= -1.0f;
        }
        else if (d_instancePositions[i].x < -1.0f)
        {
            d_instancePositions[i].x = -1.0f;
            d_instanceVelocities[i].x *= -1.0f;
        }

        if (d_instancePositions[i].y > 1.0f)
        {
            d_instancePositions[i].y = 1.0f;
            d_instanceVelocities[i].y *= -1.0f;
        }
        else if (d_instancePositions[i].y < -1.0f)
        {
            d_instancePositions[i].y = -1.0f;
            d_instanceVelocities[i].y *= -1.0f;
        }

        // Get max velocity
        float maxVelocity = 0.0f;
        for (int j = 0; j < numParticles; ++j)
        {
            if (norm(d_instanceVelocities[i]) > maxVelocity)
            {
                maxVelocity = norm(d_instanceVelocities[i]);
            }
        }

        d_instanceColors[i] = make_float4(
            // norm(d_instanceVelocities[i]) / maxVelocity, // r
            (d_instancePositions[i].x + 1.0) / 2.0, // r
            1.0f,                     // g
            1.0f,                     // b
            1.0f);                    // a
    }
}

int main()
{
    glfwSetErrorCallback(glfwErrorCallback);
    if (!glfwInit())
        return -1;

    GLFWwindow *window = glfwCreateWindow(WIDTH, HEIGHT, "CUDA Particles", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    // set callback functions
    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetScrollCallback(window, scroll_callback);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // Initialize GLAD2
    int version = gladLoadGL(glfwGetProcAddress);

    // Print version info
    std::cout << "GLAD2 GL version: " << GLAD_VERSION_MAJOR(version) << "." << GLAD_VERSION_MINOR(version) << std::endl;
    std::cout << "OpenGL version: " << glGetString(GL_VERSION) << std::endl;
    std::cout << "Renderer: " << glGetString(GL_RENDERER) << std::endl;
    std::cout << "Vendor: " << glGetString(GL_VENDOR) << std::endl;

    glViewport(0, 0, WIDTH, HEIGHT);

    Shader shader("shaders/particles.vertex.glsl", "shaders/particles.geometry.glsl", "shaders/particles.fragment.glsl");

    uint numParticles = 50000;
    std::cout << "Particle count: " << numParticles << std::endl;

    // Particle positions
    std::vector<float3> instancePositionData;
    for (int i = 0; i < numParticles; i++)
    {
        float3 position = make_float3(
            (float)rand() / RAND_MAX - 0.5f, // x
            (float)rand() / RAND_MAX - 0.5f, // y
            0.0f);                           // z
        instancePositionData.push_back(position);
    }

    // Particle velocities
    std::vector<float3> instanceVelocityData;
    for (int i = 0; i < numParticles; i++)
    {
        float3 velocity = make_float3(
            (2.0f * rand() / RAND_MAX - 1.0f) * 0.005f, // x
            (2.0f * rand() / RAND_MAX - 1.0f) * 0.005f, // y
            0.0f);                                      // z
        instanceVelocityData.push_back(velocity);
    }

    // Particle colors
    std::vector<float4> instanceColorData;
    for (int i = 0; i < numParticles; i++)
    {
        float4 color = make_float4(
            (2.0f * rand() / RAND_MAX - 1.0f) * 0.005f, // r
            (2.0f * rand() / RAND_MAX - 1.0f) * 0.005f, // g
            (2.0f * rand() / RAND_MAX - 1.0f) * 0.005f, // b
            1.0f);                                      // a
        instanceColorData.push_back(color);
    }

    // Particle VAO (for instanced rendering)
    GLuint particleVao;
    glGenVertexArrays(1, &particleVao);
    glBindVertexArray(particleVao);

    // Instance positions
    GLuint instancePositionsVbo;
    glGenBuffers(1, &instancePositionsVbo);
    glBindBuffer(GL_ARRAY_BUFFER, instancePositionsVbo);
    glBufferData(
        GL_ARRAY_BUFFER,               // target
        numParticles * sizeof(float3), // size
        instancePositionData.data(),   // data
        GL_STATIC_DRAW);               // usage

    // Position attribute
    glVertexAttribPointer(
        0,                 // index
        3,                 // size
        GL_FLOAT,          // type
        GL_FALSE,          // normalized
        3 * sizeof(float), // stride
        (void *)0);        // pointer
    glEnableVertexAttribArray(0);
    glVertexAttribDivisor(0, 1); // Update attribute every 1 instance

    // Instance velocities
    GLuint instanceVelocitiesVbo;
    glGenBuffers(1, &instanceVelocitiesVbo);
    glBindBuffer(GL_ARRAY_BUFFER, instanceVelocitiesVbo);
    glBufferData(
        GL_ARRAY_BUFFER,               // target
        numParticles * sizeof(float3), // size
        instanceVelocityData.data(),   // data
        GL_STATIC_DRAW);               // usage

    // Velocity attribute
    glVertexAttribPointer(
        1,                 // index
        3,                 // size
        GL_FLOAT,          // type
        GL_FALSE,          // normalized
        3 * sizeof(float), // stride
        (void *)0);        // pointer
    glEnableVertexAttribArray(1);
    glVertexAttribDivisor(1, 1); // Update attribute every 1 instance

    // Instance colors
    GLuint instanceColorsVbo;
    glGenBuffers(1, &instanceColorsVbo);
    glBindBuffer(GL_ARRAY_BUFFER, instanceColorsVbo);
    glBufferData(
        GL_ARRAY_BUFFER,               // target
        numParticles * sizeof(float4), // size
        instanceColorData.data(),      // data
        GL_STATIC_DRAW);               // usage

    // Color attribute
    glVertexAttribPointer(
        2,                 // index
        4,                 // size
        GL_FLOAT,          // type
        GL_FALSE,          // normalized
        4 * sizeof(float), // stride
        (void *)0);        // pointer
    glEnableVertexAttribArray(2);
    glVertexAttribDivisor(2, 1); // Update attribute every 1 instance

    glBindBuffer(GL_ARRAY_BUFFER, 0); // Unbind VBO
    glBindVertexArray(0);             // Unbind VAO

    // Register instance VBOs with CUDA
    hipGraphicsResource *cuda_positions_vbo_resource;
    hipGraphicsResource *cuda_velocities_vbo_resource;
    hipGraphicsResource *cuda_colors_vbo_resource;
    hipGraphicsGLRegisterBuffer(&cuda_positions_vbo_resource, instancePositionsVbo, cudaGraphicsMapFlagsWriteDiscard);
    hipGraphicsGLRegisterBuffer(&cuda_velocities_vbo_resource, instanceVelocitiesVbo, cudaGraphicsMapFlagsWriteDiscard);
    hipGraphicsGLRegisterBuffer(&cuda_colors_vbo_resource, instanceColorsVbo, cudaGraphicsMapFlagsWriteDiscard);

    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    while (!glfwWindowShouldClose(window))
    {
        glClearColor(0.15f, 0.15f, 0.15f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // Map instance VBOs to CUDA on device
        float3 *d_instancePositions;
        float3 *d_instanceVelocities;
        float4 *d_instanceColors;
        hipGraphicsMapResources(1, &cuda_positions_vbo_resource, 0);
        hipGraphicsMapResources(1, &cuda_velocities_vbo_resource, 0);
        hipGraphicsMapResources(1, &cuda_colors_vbo_resource, 0);
        hipGraphicsResourceGetMappedPointer((void **)&d_instancePositions, nullptr, cuda_positions_vbo_resource);
        hipGraphicsResourceGetMappedPointer((void **)&d_instanceVelocities, nullptr, cuda_velocities_vbo_resource);
        hipGraphicsResourceGetMappedPointer((void **)&d_instanceColors, nullptr, cuda_colors_vbo_resource);

        // Update particles
        int threadsPerBlock = 256;
        int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
        updateParticles<<<blocksPerGrid, threadsPerBlock>>>(d_instancePositions, d_instanceVelocities, d_instanceColors, numParticles, glfwGetTime());
        hipError_t err = hipDeviceSynchronize();
        err = hipDeviceSynchronize(); // Blocks execution until kernel is finished
        if (err != hipSuccess)
        {
            std::cerr << "Failed to synchronize on the CUDA device (error code " << hipGetErrorString(err) << ")!" << std::endl;
            exit(EXIT_FAILURE);
        }

        // Unmap instance VBO data from CUDA (so OpenGL can use it)
        hipGraphicsUnmapResources(1, &cuda_positions_vbo_resource, 0);
        hipGraphicsUnmapResources(1, &cuda_velocities_vbo_resource, 0);
        hipGraphicsUnmapResources(1, &cuda_colors_vbo_resource, 0);

        // Draw particles
        shader.use();
        glBindVertexArray(particleVao);
        glDrawArraysInstanced(
            GL_POINTS,     // mode: type of primitives to render
            0,             // first: starting index
            1,             // count: number of indicies to render
            numParticles); // instance count
        glBindVertexArray(0);

        handleInput(window);
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Clean up
    hipGraphicsUnregisterResource(cuda_positions_vbo_resource);
    hipGraphicsUnregisterResource(cuda_velocities_vbo_resource);
    hipGraphicsUnregisterResource(cuda_colors_vbo_resource);
    glDeleteVertexArrays(1, &particleVao);
    glDeleteBuffers(1, &instancePositionsVbo);
    glDeleteBuffers(1, &instanceVelocitiesVbo);
    glfwTerminate();
    return 0;
}

void glfwErrorCallback(int error, const char *description)
{
    std::cerr << "GLFW Error: " << error << " - " << description << std::endl;
}

void handleInput(GLFWwindow *window)
{
    // exit program
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
    {
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    }
}

void mouse_callback(GLFWwindow *window, double xPos, double yPos)
{
    // TODO Implement
    // std::cout << "Mouse position: (" << xPos << ", " << yPos << ")" << std::endl;
}

void scroll_callback(GLFWwindow *window, double xOffset, double yOffset)
{
    std::cout << "Scroll offset: (" << xOffset << ", " << yOffset << ")" << std::endl;
}

void framebuffer_size_callback(GLFWwindow *window, int newWidth, int newHeight)
{
    glViewport(0, 0, newWidth, newHeight);
}
