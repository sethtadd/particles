#include "CudaUnifiedMemory.hpp"
#include <hip/hip_runtime.h>

CudaUnifiedMemory::CudaUnifiedMemory(std::size_t size) : size_(size)
{
    hipMallocManaged(&pointer_, size_);
}

CudaUnifiedMemory::~CudaUnifiedMemory()
{
    hipFree(pointer_);
}

void *CudaUnifiedMemory::getPointer()
{
    return pointer_;
}
